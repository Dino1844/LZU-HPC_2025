#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>
#include <chrono>
#include "utils.h"
#include "reduce.h"
#include <hip/hip_runtime.h>


void test_reduce(const int len,const int iter_time){
    float * a =(float*)malloc(len * sizeof(float));
    Gen_Matrix<float>(a,len);
    float sum=0.0f;
    #pragma omp parallel for reduction(+:sum) schedule(static,1024)
    for(int i=0;i<len;i++){
        sum+=a[i];
    }
    // printf("CPU sum is %f\n",sum);
    double min_time=1e6;
    float max_diff=0.0f;
    float* d_in;
    hipMalloc(&d_in, len * sizeof(float));
    hipMemcpy(d_in,a,len*sizeof(float),hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    for(int i=0;i<iter_time;i++){
        auto iter_start = std::chrono::high_resolution_clock::now();
        float gpu_sum=gpuReduce(d_in, len);
        auto iter_end = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(iter_end - iter_start);
        if(std::abs(gpu_sum - sum) > max_diff){
            max_diff = std::abs(gpu_sum - sum);
        }
        min_time = std::min(duration.count() / 1e3,min_time); 
    }
    std::cout<<"len: "<<len<<" , time: "<<min_time<<" us"<<std::endl;
    if(max_diff>sum*1e-5){
        std::cout<<"Result incorrect! diff is "<<max_diff<<std::endl;
        free(a);
        return;
    }else{
        std::cout<<"Result correct! diff is "<<max_diff <<std::endl;
    }
    hipFree(d_in);
    free(a);

}